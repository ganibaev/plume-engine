#include "tcnn_wrapper.h"
#include <stdio.h>

#include <tiny-cuda-nn/common.h>
#include <tiny-cuda-nn/gpu_matrix.h>
#include <tiny-cuda-nn/config.h>
#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/multi_stream.h>

#define TRAINING_BATCH_SIZE (1 << 17)
#define INFERENCE_BATCH_SIZE (1920 * 1080)
#define TRAINING_STEPS_PER_FRAME 1


struct RadianceCache
{
	hipStream_t trainingStream;
	hipStream_t inferenceStream;
	uint32_t inputDims;
	uint32_t outputDims;
    tcnn::TrainableModel model;

	float* trainingDataBuffer = nullptr;
	float* trainingTargetsBuffer = nullptr;
	float* queryBuffer = nullptr;

	float* resBuffer = nullptr;
};

static RadianceCache diffuseRC;
static RadianceCache specularRC;
static RadianceCache classicRC;


void vktcnn::create_cache_from_config(uint32_t inputDims, uint32_t outputDims, nlohmann::json config, vktcnn::CacheType type)
{
	auto model = tcnn::create_from_config(inputDims, outputDims, config);

	switch (type)
	{
	case vktcnn::CacheType::eDiffuse:
		diffuseRC.model = model;
		diffuseRC.inputDims = inputDims;
		diffuseRC.outputDims = outputDims;

		CUDA_CHECK_THROW(hipStreamCreate(&diffuseRC.inferenceStream));
		diffuseRC.trainingStream = diffuseRC.inferenceStream;
		break;
	case vktcnn::CacheType::eSpecular:
		specularRC.model = model;
		specularRC.inputDims = inputDims;
		specularRC.outputDims = outputDims;

		CUDA_CHECK_THROW(hipStreamCreate(&specularRC.inferenceStream));
		specularRC.trainingStream = specularRC.inferenceStream;
		break;
	case vktcnn::CacheType::eClassic:
		classicRC.model = model;
		classicRC.inputDims = inputDims;
		classicRC.outputDims = outputDims;

		CUDA_CHECK_THROW(hipStreamCreate(&classicRC.inferenceStream));
		classicRC.trainingStream = classicRC.inferenceStream;
		break;
	default:
		break;
	}
}

float* vktcnn::get_external_memory_ptr(HANDLE handle, uint64_t size)
{
	hipExternalMemory_t ext = nullptr;

	hipExternalMemoryHandleDesc desc = {};
	desc.type = hipExternalMemoryHandleTypeOpaqueWin32;
	desc.handle.win32.handle = handle;
	desc.size = size;

	CUDA_CHECK_THROW(hipImportExternalMemory(&ext, &desc));

	void* ptr = nullptr;

	hipExternalMemoryBufferDesc bDesc = {};
	bDesc.flags = 0;
	bDesc.offset = 0;
	bDesc.size = size;

	CUDA_CHECK_THROW(hipExternalMemoryGetMappedBuffer(&ptr, ext, &bDesc));

	return reinterpret_cast<float*>(ptr);
}

void vktcnn::train(uint32_t numElements, const HANDLE inputs, const HANDLE targets, CacheType cacheType,
	size_t inputBufferSize, size_t targetBufferSize)
{
	RadianceCache* cache = nullptr;
	switch (cacheType)
	{
	case vktcnn::CacheType::eDiffuse:
		cache = &diffuseRC;
		break;
	case vktcnn::CacheType::eSpecular:
		cache = &specularRC;
		break;
	case vktcnn::CacheType::eClassic:
		cache = &classicRC;
		break;
	default:
		return;
	}

	if (numElements < tcnn::batch_size_granularity * TRAINING_STEPS_PER_FRAME)
	{
		return;
	}

	if (!cache->trainingDataBuffer)
	{
		cache->trainingDataBuffer = get_external_memory_ptr(inputs, inputBufferSize);
	}

	if (!cache->trainingTargetsBuffer)
	{
		cache->trainingTargetsBuffer = get_external_memory_ptr(targets, targetBufferSize);
	}

	uint32_t largestBatch = tcnn::previous_multiple(numElements, tcnn::batch_size_granularity * TRAINING_STEPS_PER_FRAME);

	uint32_t splitBatchSize = largestBatch / TRAINING_STEPS_PER_FRAME;
	CUDA_CHECK_THROW(hipDeviceSynchronize());
	for (int i = 0; i < TRAINING_STEPS_PER_FRAME; ++i)
	{
		tcnn::GPUMatrix<float> inputMatrix(cache->trainingDataBuffer + splitBatchSize * i, cache->inputDims, splitBatchSize);
		tcnn::GPUMatrix<float> targetMatrix(cache->trainingTargetsBuffer + splitBatchSize * i, cache->outputDims, splitBatchSize);

		tcnn::SyncedMultiStream syncedStream{ cache->trainingStream, 2 };
		auto ctx = cache->model.trainer->training_step(syncedStream.get(1), inputMatrix, targetMatrix);
	}
	CUDA_CHECK_THROW(hipDeviceSynchronize());
}

void vktcnn::inference(uint32_t numElements, const HANDLE input, HANDLE output, CacheType cacheType, size_t inputBufferSize,
	size_t resBufferSize)
{
	RadianceCache* cache = nullptr;
	switch (cacheType)
	{
	case vktcnn::CacheType::eDiffuse:
		cache = &diffuseRC;
		break;
	case vktcnn::CacheType::eSpecular:
		cache = &specularRC;
		break;
	case vktcnn::CacheType::eClassic:
		cache = &classicRC;
		break;
	default:
		return;
	}

	if (!cache->queryBuffer)
	{
		cache->queryBuffer = get_external_memory_ptr(input, inputBufferSize);
	}

	if (!cache->resBuffer)
	{
		cache->resBuffer = get_external_memory_ptr(output, resBufferSize);
	}

	tcnn::GPUMatrix<float> inputs(cache->queryBuffer, cache->inputDims, numElements);
	tcnn::GPUMatrix<float> outputs(cache->resBuffer, cache->outputDims, numElements);

	tcnn::SyncedMultiStream syncedStream{ cache->inferenceStream, 2 };
	CUDA_CHECK_THROW(hipDeviceSynchronize());
	cache->model.network->inference(syncedStream.get(1), inputs, outputs);
	CUDA_CHECK_THROW(hipDeviceSynchronize());
}

void vktcnn::terminate()
{
	tcnn::free_all_gpu_memory_arenas();
}