#include "tcnn_wrapper.h"
#include <stdio.h>

#include <tiny-cuda-nn/common.h>
#include <tiny-cuda-nn/gpu_matrix.h>
#include <tiny-cuda-nn/config.h>
#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/multi_stream.h>

#define TRAINING_BATCH_SIZE (1 << 17)
#define INFERENCE_BATCH_SIZE (1920 * 1080)
#define TRAINING_STEPS_PER_FRAME 1


struct RadianceCache
{
	hipStream_t trainingStream;
	hipStream_t inferenceStream;
	uint32_t inputDims;
	uint32_t outputDims;
    tcnn::TrainableModel model;
};

static RadianceCache diffuseRC;
static RadianceCache specularRC;
static RadianceCache classicRC;


void vktcnn::create_cache_from_config(uint32_t inputDims, uint32_t outputDims, nlohmann::json config, vktcnn::CacheType type)
{
	auto model = tcnn::create_from_config(inputDims, outputDims, config);

	switch (type)
	{
	case vktcnn::CacheType::eDiffuse:
		diffuseRC.model = model;
		diffuseRC.inputDims = inputDims;
		diffuseRC.outputDims = outputDims;

		CUDA_CHECK_THROW(hipStreamCreate(&diffuseRC.inferenceStream));
		diffuseRC.trainingStream = diffuseRC.inferenceStream;
		break;
	case vktcnn::CacheType::eSpecular:
		specularRC.model = model;
		specularRC.inputDims = inputDims;
		specularRC.outputDims = outputDims;

		CUDA_CHECK_THROW(hipStreamCreate(&specularRC.inferenceStream));
		specularRC.trainingStream = specularRC.inferenceStream;
		break;
	case vktcnn::CacheType::eClassic:
		classicRC.model = model;
		classicRC.inputDims = inputDims;
		classicRC.outputDims = outputDims;

		CUDA_CHECK_THROW(hipStreamCreate(&classicRC.inferenceStream));
		classicRC.trainingStream = classicRC.inferenceStream;
		break;
	default:
		break;
	}
}

void vktcnn::train(uint32_t numElements, const float* inputs, const float* targets, CacheType cacheType)
{
	RadianceCache* cache = nullptr;
	switch (cacheType)
	{
	case vktcnn::CacheType::eDiffuse:
		cache = &diffuseRC;
		break;
	case vktcnn::CacheType::eSpecular:
		cache = &specularRC;
		break;
	case vktcnn::CacheType::eClassic:
		cache = &classicRC;
		break;
	default:
		break;
	}

	if (numElements < tcnn::batch_size_granularity * TRAINING_STEPS_PER_FRAME)
	{
		return;
	}

	tcnn::GPUMemory<float> inputMemory(numElements * cache->inputDims);
	tcnn::GPUMemory<float> targetMemory(numElements * cache->outputDims);

	inputMemory.copy_from_host(inputs);
	targetMemory.copy_from_host(targets);

	uint32_t largestBatch = tcnn::previous_multiple(numElements, tcnn::batch_size_granularity * TRAINING_STEPS_PER_FRAME);

	uint32_t splitBatchSize = largestBatch / TRAINING_STEPS_PER_FRAME;

	for (int i = 0; i < TRAINING_STEPS_PER_FRAME; ++i)
	{
		tcnn::GPUMatrix<float> inputMatrix(inputMemory.data() + splitBatchSize * i, cache->inputDims, splitBatchSize);
		tcnn::GPUMatrix<float> targetMatrix(targetMemory.data() + splitBatchSize * i, cache->outputDims, splitBatchSize);

		tcnn::SyncedMultiStream syncedStream{ cache->trainingStream, 2 };
		auto ctx = cache->model.trainer->training_step(syncedStream.get(1), inputMatrix, targetMatrix);
	}
}

void vktcnn::inference(uint32_t numElements, const float* input, float* output, CacheType cacheType)
{
	RadianceCache* cache = nullptr;
	switch (cacheType)
	{
	case vktcnn::CacheType::eDiffuse:
		cache = &diffuseRC;
		break;
	case vktcnn::CacheType::eSpecular:
		cache = &specularRC;
		break;
	case vktcnn::CacheType::eClassic:
		cache = &classicRC;
		break;
	default:
		break;
	}

	tcnn::GPUMemory<float> inputMemory(numElements * cache->inputDims);
	inputMemory.copy_from_host(input);
	tcnn::GPUMemory<float> outputMemory(numElements * cache->outputDims);


	tcnn::GPUMatrix<float> inputs(inputMemory.data(), cache->inputDims, numElements);
	tcnn::GPUMatrix<float> outputs(outputMemory.data(), cache->outputDims, numElements);

	tcnn::SyncedMultiStream syncedStream{ cache->inferenceStream, 2 };
	cache->model.network->inference(syncedStream.get(1), inputs, outputs);
	hipDeviceSynchronize();
	outputMemory.copy_to_host(output);
}

void vktcnn::terminate()
{
	tcnn::free_all_gpu_memory_arenas();
}